#include "hip/hip_runtime.h"
#include <>
#include "taichi/ui/backend/vulkan/vertex.h"

int div_up(int a, int b) {
  if (b == 0) {
    return 1;
  }
  int result = (a % b != 0) ? (a / b + 1) : (a / b);
  return result;
}

#define MAX_THREADS_PER_BLOCK 1024
void set_num_blocks_threads(int N, int &num_blocks, int &num_threads) {
  num_threads = min(N, MAX_THREADS_PER_BLOCK);
  num_blocks = div_up(N, num_threads);
}

__global__ void update_renderables_vertices_cuda_impl(Vertex *vbo,
                                                      float *vertices,
                                                      int num_vertices,
                                                      int num_components) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i >= num_vertices)
    return;

  vbo[i].pos.x = vertices[i * num_components];
  vbo[i].pos.y = vertices[i * num_components + 1];
  if (num_components == 3) {
    vbo[i].pos.z = vertices[i * num_components + 2];
  }
}

void update_renderables_vertices_cuda(Vertex *vbo,
                                      float *vertices,
                                      int num_vertices,
                                      int num_components) {
  int num_blocks, num_threads;
  set_num_blocks_threads(num_vertices, num_blocks, num_threads);
  update_renderables_vertices_cuda_impl<<<num_blocks, num_threads>>>(
      vbo, vertices, num_vertices, num_components);
}

void update_renderables_vertices_x64(Vertex *vbo,
                                     float *vertices,
                                     int num_vertices,
                                     int num_components) {
  for (int i = 0; i < num_vertices; ++i) {
    vbo[i].pos.x = vertices[i * num_components];
    vbo[i].pos.y = vertices[i * num_components + 1];
    if (num_components == 3) {
      vbo[i].pos.z = vertices[i * num_components + 2];
    }
  }
}

__global__ void update_renderables_indices_cuda_impl(int *ibo,
                                                     int *indices,
                                                     int num_indices) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i >= num_indices)
    return;

  ibo[i] = indices[i];
}
void update_renderables_indices_cuda(int *ibo, int *indices, int num_indices) {
  int num_blocks, num_threads;
  set_num_blocks_threads(num_indices, num_blocks, num_threads);
  update_renderables_indices_cuda_impl<<<num_blocks, num_threads>>>(
      ibo, indices, num_indices);
}

void update_renderables_indices_x64(int *ibo, int *indices, int num_indices) {
  for (int i = 0; i < num_indices; ++i) {
    ibo[i] = indices[i];
  }
}

__global__ void update_renderables_indices_unindexed_cuda_impl(
    int *ibo,
    int num_indices) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i >= num_indices)
    return;
  ibo[i] = i;
}
void update_renderables_indices_unindexed_cuda(int *ibo, int num_indices) {
  int num_blocks, num_threads;
  set_num_blocks_threads(num_indices, num_blocks, num_threads);
  update_renderables_indices_unindexed_cuda_impl<<<num_blocks, num_threads>>>(
      ibo, num_indices);
}
void update_renderables_indices_unindexed_x64(int *ibo, int num_indices) {
  for (int i = 0; i < num_indices; ++i) {
    ibo[i] = i;
  }
}

__global__ void update_renderables_colors_cuda_impl(Vertex *vbo,
                                                    float *colors,
                                                    int num_vertices) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i >= num_vertices)
    return;

  vbo[i].color.x = colors[i * 3];
  vbo[i].color.y = colors[i * 3 + 1];
  vbo[i].color.z = colors[i * 3 + 2];
}
void update_renderables_colors_cuda(Vertex *vbo,
                                    float *colors,
                                    int num_vertices) {
  int num_blocks, num_threads;
  set_num_blocks_threads(num_vertices, num_blocks, num_threads);
  update_renderables_colors_cuda_impl<<<num_blocks, num_threads>>>(
      vbo, colors, num_vertices);
}

void update_renderables_colors_x64(Vertex *vbo,
                                   float *colors,
                                   int num_vertices) {
  for (int i = 0; i < num_vertices; ++i) {
    vbo[i].color.x = colors[i * 3];
    vbo[i].color.y = colors[i * 3 + 1];
    vbo[i].color.z = colors[i * 3 + 2];
  }
}

__global__ void update_renderables_normals_cuda_impl(Vertex *vbo,
                                                     float *normals,
                                                     int num_vertices) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i >= num_vertices)
    return;

  vbo[i].normal.x = normals[i * 3];
  vbo[i].normal.y = normals[i * 3 + 1];
  vbo[i].normal.z = normals[i * 3 + 2];
}
void update_renderables_normals_cuda(Vertex *vbo,
                                     float *normals,
                                     int num_vertices) {
  int num_blocks, num_threads;
  set_num_blocks_threads(num_vertices, num_blocks, num_threads);
  update_renderables_normals_cuda_impl<<<num_blocks, num_threads>>>(
      vbo, normals, num_vertices);
}
void update_renderables_normals_x64(Vertex *vbo,
                                    float *normals,
                                    int num_vertices) {
  for (int i = 0; i < num_vertices; ++i) {
    vbo[i].normal.x = normals[i * 3];
    vbo[i].normal.y = normals[i * 3 + 1];
    vbo[i].normal.z = normals[i * 3 + 2];
  }
}

// We implement lines by generating rectangles. Note there this requires careful
// treatment of aspect ratios.

__global__ void update_lines_vbo_cuda_impl(Vertex *vbo,
                                           int *ibo,
                                           float *vertices,
                                           int N,
                                           float width,
                                           float aspect_ratio,
                                           float *colors,
                                           bool use_per_vertex_color) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i >= N)
    return;
  float a_x = vertices[i * 4];
  float a_y = vertices[i * 4 + 1];
  float b_x = vertices[i * 4 + 2];
  float b_y = vertices[i * 4 + 3];

  float distance =
      sqrt((a_x - b_x) * (a_x - b_x) * aspect_ratio * aspect_ratio +
           (a_y - b_y) * (a_y - b_y));
  float factor = (width / 2.f) / distance;
  float dx = (b_y - a_y) * factor / aspect_ratio;
  float dy = (b_x - a_x) * aspect_ratio * factor;

  vbo[i * 4].pos.x = a_x + dx;
  vbo[i * 4].pos.y = a_y - dy;
  vbo[i * 4].pos.z = 0;

  vbo[i * 4 + 1].pos.x = a_x - dx;
  vbo[i * 4 + 1].pos.y = a_y + dy;
  vbo[i * 4 + 1].pos.z = 0;

  vbo[i * 4 + 2].pos.x = b_x + dx;
  vbo[i * 4 + 2].pos.y = b_y - dy;
  vbo[i * 4 + 2].pos.z = 0;

  vbo[i * 4 + 3].pos.x = b_x - dx;
  vbo[i * 4 + 3].pos.y = b_y + dy;
  vbo[i * 4 + 3].pos.z = 0;

  if (use_per_vertex_color) {
    float3 color_a = ((float3 *)colors)[i * 2];
    float3 color_b = ((float3 *)colors)[i * 2 + 1];

    vbo[i * 4].color.x = color_a.x;
    vbo[i * 4].color.y = color_a.y;
    vbo[i * 4].color.z = color_a.z;

    vbo[i * 4 + 1].color.x = color_a.x;
    vbo[i * 4 + 1].color.y = color_a.y;
    vbo[i * 4 + 1].color.z = color_a.z;

    vbo[i * 4 + 2].color.x = color_b.x;
    vbo[i * 4 + 2].color.y = color_b.y;
    vbo[i * 4 + 2].color.z = color_b.z;

    vbo[i * 4 + 3].color.x = color_b.x;
    vbo[i * 4 + 3].color.y = color_b.y;
    vbo[i * 4 + 3].color.z = color_b.z;
  }

  ibo[i * 6] = i * 4;
  ibo[i * 6 + 1] = i * 4 + 1;
  ibo[i * 6 + 2] = i * 4 + 2;

  ibo[i * 6 + 3] = i * 4 + 1;
  ibo[i * 6 + 4] = i * 4 + 2;
  ibo[i * 6 + 5] = i * 4 + 3;
}
void update_lines_vbo_cuda(Vertex *vbo,
                           int *ibo,
                           float *vertices,
                           int N,
                           float width,
                           float aspect_ratio,
                           float *colors,
                           bool use_per_vertex_color) {
  int num_blocks, num_threads;
  set_num_blocks_threads(N, num_blocks, num_threads);
  update_lines_vbo_cuda_impl<<<num_blocks, num_threads>>>(
      vbo, ibo, vertices, N, width, aspect_ratio, colors, use_per_vertex_color);
}
void update_lines_vbo_x64(Vertex *vbo,
                          int *ibo,
                          float *vertices,
                          int N,
                          float width,
                          float aspect_ratio,
                          float *colors,
                          bool use_per_vertex_color) {
  for (int i = 0; i < N; ++i) {
    float a_x = vertices[i * 4];
    float a_y = vertices[i * 4 + 1];
    float b_x = vertices[i * 4 + 2];
    float b_y = vertices[i * 4 + 3];

    float distance =
        sqrt((a_x - b_x) * (a_x - b_x) * aspect_ratio * aspect_ratio +
             (a_y - b_y) * (a_y - b_y));
    float factor = (width / 2.f) / distance;
    float dx = (b_y - a_y) * factor / aspect_ratio;
    float dy = (b_x - a_x) * aspect_ratio * factor;

    vbo[i * 4].pos.x = a_x + dx;
    vbo[i * 4].pos.y = a_y - dy;
    vbo[i * 4].pos.z = 0;

    vbo[i * 4 + 1].pos.x = a_x - dx;
    vbo[i * 4 + 1].pos.y = a_y + dy;
    vbo[i * 4 + 1].pos.z = 0;

    vbo[i * 4 + 2].pos.x = b_x + dx;
    vbo[i * 4 + 2].pos.y = b_y - dy;
    vbo[i * 4 + 2].pos.z = 0;

    vbo[i * 4 + 3].pos.x = b_x - dx;
    vbo[i * 4 + 3].pos.y = b_y + dy;
    vbo[i * 4 + 3].pos.z = 0;

    if (use_per_vertex_color) {
      float3 color_a = ((float3 *)colors)[i * 2];
      float3 color_b = ((float3 *)colors)[i * 2 + 1];

      vbo[i * 4].color.x = color_a.x;
      vbo[i * 4].color.y = color_a.y;
      vbo[i * 4].color.z = color_a.z;

      vbo[i * 4 + 1].color.x = color_a.x;
      vbo[i * 4 + 1].color.y = color_a.y;
      vbo[i * 4 + 1].color.z = color_a.z;

      vbo[i * 4 + 2].color.x = color_b.x;
      vbo[i * 4 + 2].color.y = color_b.y;
      vbo[i * 4 + 2].color.z = color_b.z;

      vbo[i * 4 + 3].color.x = color_b.x;
      vbo[i * 4 + 3].color.y = color_b.y;
      vbo[i * 4 + 3].color.z = color_b.z;
    }

    ibo[i * 6] = i * 4;
    ibo[i * 6 + 1] = i * 4 + 1;
    ibo[i * 6 + 2] = i * 4 + 2;

    ibo[i * 6 + 3] = i * 4 + 1;
    ibo[i * 6 + 4] = i * 4 + 2;
    ibo[i * 6 + 5] = i * 4 + 3;
  }
}

template <typename T>
__device__ __host__ inline unsigned char get_color_value(T x);

template <>
__device__ __host__ inline unsigned char get_color_value<unsigned char>(
    unsigned char x) {
  return x;
}

template <>
__device__ __host__ inline unsigned char get_color_value<float>(float x) {
  x = max(0.f, min(1.f, x));
  return (unsigned char)(x * 255);
}

template <typename T>
__global__ void copy_to_texture_fuffer_cuda_impl(T *src,
                                                 uint64_t surface,
                                                 int width,
                                                 int height,
                                                 int actual_width,
                                                 int actual_height,
                                                 int channels) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i >= width * height)
    return;

  int y = i / width;
  int x = i % width;

  T *src_base_addr = src + (x * actual_height + y) * channels;
  uchar4 data = make_uchar4(0, 0, 0, 0);

  data.x = get_color_value<T>(src_base_addr[0]);
  data.y = get_color_value<T>(src_base_addr[1]);
  data.z = get_color_value<T>(src_base_addr[2]);
  data.w = 255;

  surf3Dwrite(data, surface, x * sizeof(uchar4), y,0);
}

template <typename T>
void copy_to_texture_fuffer_cuda(T *src,
                                 uint64_t surface,
                                 int width,
                                 int height,
                                 int actual_width,
                                 int actual_height,
                                 int channels) {
  int num_blocks, num_threads;
  set_num_blocks_threads(width * height, num_blocks, num_threads);
  copy_to_texture_fuffer_cuda_impl<<<num_blocks, num_threads>>>(
      src, (uint64_t)surface, width, height, actual_width, actual_height,
      channels);
}

template <typename T>
void copy_to_texture_fuffer_x64(T *src,
                                unsigned char *dest,
                                int width,
                                int height,
                                int actual_width,
                                int actual_height,
                                int channels) {
  for (int i = 0; i < width * height; ++i) {
    int y = i / width;
    int x = i % width;

    T *src_base_addr = src + (x * actual_height + y) * channels;
    uchar4 data = make_uchar4(0, 0, 0, 0);

    data.x = get_color_value<T>(src_base_addr[0]);
    data.y = get_color_value<T>(src_base_addr[1]);
    data.z = get_color_value<T>(src_base_addr[2]);
    data.w = 255;

    ((uchar4 *)dest)[y * width + x] = data;
  }
}

template void copy_to_texture_fuffer_cuda<float>(float *src,
                                                 uint64_t surface,
                                                 int width,
                                                 int height,
                                                 int actual_width,
                                                 int actual_height,
                                                 int channels);
template void copy_to_texture_fuffer_cuda<unsigned char>(unsigned char *src,
                                                         uint64_t surface,
                                                         int width,
                                                         int height,
                                                         int actual_width,
                                                         int actual_height,
                                                         int channels);

template void copy_to_texture_fuffer_x64<float>(float *src,
                                                unsigned char *dest,
                                                int width,
                                                int height,
                                                int actual_width,
                                                int actual_height,
                                                int channels);
template void copy_to_texture_fuffer_x64<unsigned char>(unsigned char *src,
                                                        unsigned char *dest,
                                                        int width,
                                                        int height,
                                                        int actual_width,
                                                        int actual_height,
                                                        int channels);
