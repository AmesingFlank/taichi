#include "hip/hip_runtime.h"
#include "set_image.h"
#include "../vulkan_cuda_interop.h"
#include "../vulkan_cuda_interop.h"
#include "../../../utils/utils.h"
#include "../../../common/constants.h"

namespace vulkan {


template<typename T>
__device__ __host__
inline unsigned char get_color_value(T x);

template<>
__device__ __host__
inline unsigned char get_color_value<unsigned char>(unsigned char x){
    return x;
}

template<>
__device__ __host__
inline unsigned char get_color_value<float>(float x){
    x = max(0.f,min(1.f,x));
    return (unsigned char)(x * 255);
}


template<typename T>
__global__
void copy_to_texture_fuffer_cuda(T* src, hipSurfaceObject_t surface, int width, int height, int actual_width, int actual_height, int channels){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= width * height) return;

    int y = i / width;
    int x = i % width;

    T* src_base_addr = src + (x*actual_height + y) * channels;
    uchar4 data = make_uchar4(0,0,0,0);
    
    data.x = get_color_value<T>(src_base_addr[0]);
    data.y = get_color_value<T>(src_base_addr[1]);
    data.z  = get_color_value<T>(src_base_addr[2]);
    data.w = 255;
    
    surf2Dwrite(data, surface, x* sizeof(uchar4), y);
}

template<typename T>
void copy_to_texture_fuffer_x64(T* src, uchar4* dest, int width, int height, int actual_width, int actual_height, int channels){
    for(int i = 0;i<width * height;++i){
        int y = i / width;
        int x = i % width;

        T* src_base_addr = src + (x*actual_height + y) * channels;
        uchar4 data = make_uchar4(0,0,0,0);
        
        data.x = get_color_value<T>(src_base_addr[0]);
        data.y = get_color_value<T>(src_base_addr[1]);
        data.z  = get_color_value<T>(src_base_addr[2]);
        data.w = 255;

        dest[y * width + x] = data;
    }
    
}

void SetImage::update_data(const SetImageInfo& info){
    const FieldInfo& img = info.img;
    if(img.shape.size() != 2){
        throw std::runtime_error("for set image, the image should have exactly two axis. e,g, ti.Vector.field(3,ti.u8,(1920,1080) ");
    }
    if( (img.matrix_rows != 3 && img.matrix_rows != 4) || img.matrix_cols != 1 ){
        throw std::runtime_error("for set image, the image should either a 3-D vector field (RGB) or a 4D vector field (RGBA) ");
    }
    int new_width  = img.shape[0];
    int new_height =  img.shape[1];

    if(new_width != width || new_height != height){
        cleanup_swap_chain();
        cleanup();
        init_set_image(app_context_,new_width,new_height);
    }
    
    int actual_width = next_power_of_2(width);
    int actual_height = next_power_of_2(height);

    int pixels = width * height;
    int num_blocks,num_threads;
    set_num_blocks_threads(pixels,num_blocks,num_threads);

    if(img.field_source == FIELD_SOURCE_CUDA){
        if(img.dtype == DTYPE_U8){
            copy_to_texture_fuffer_cuda<<<num_blocks,num_threads>>>((unsigned char*)img.data,(hipSurfaceObject_t)texture_surface_,width,height,actual_width,actual_height,img.matrix_rows);
        }
        else if (img.dtype == DTYPE_F32){
            copy_to_texture_fuffer_cuda<<<num_blocks,num_threads>>>((float*)img.data,(hipSurfaceObject_t)texture_surface_,width,height,actual_width,actual_height,img.matrix_rows);
        }
        else{
            throw std::runtime_error("for set image, dtype must be u8 or f32");
        }
        CHECK_CUDA_ERROR("copy to texture\n");
    }
    else if(img.field_source == FIELD_SOURCE_X64){
        transition_image_layout(texture_image_, VK_FORMAT_R8G8B8A8_SRGB, VK_IMAGE_LAYOUT_SHADER_READ_ONLY_OPTIMAL,VK_IMAGE_LAYOUT_TRANSFER_DST_OPTIMAL,app_context_->command_pool,app_context_->device,app_context_->graphics_queue);
        
        MappedMemory mapped_buffer(app_context_->device, staging_buffer_memory_ , pixels * sizeof(uchar4));
 
        if(img.dtype == DTYPE_U8){
            copy_to_texture_fuffer_x64 ((unsigned char*)img.data,(uchar4*)mapped_buffer.data,width,height,actual_width,actual_height,img.matrix_rows);
        }
        else if (img.dtype == DTYPE_F32){
            copy_to_texture_fuffer_x64((float*)img.data,(uchar4*)mapped_buffer.data,width,height,actual_width,actual_height,img.matrix_rows);
        }
        else{
            throw std::runtime_error("for set image, dtype must be u8 or f32");
        } 

        copy_buffer_to_image(staging_buffer_, texture_image_, width,height,app_context_->command_pool,app_context_->device,app_context_->graphics_queue);

        transition_image_layout(texture_image_, VK_FORMAT_R8G8B8A8_SRGB, VK_IMAGE_LAYOUT_TRANSFER_DST_OPTIMAL, VK_IMAGE_LAYOUT_SHADER_READ_ONLY_OPTIMAL,app_context_->command_pool,app_context_->device,app_context_->graphics_queue);
    }
    else{
        throw std::runtime_error("unsupported field source");
    }

    
}


SetImage::SetImage(AppContext* app_context){
    init_set_image(app_context,1,1);
}

void SetImage::init_set_image(AppContext* app_context,int img_width, int img_height){
    RenderableConfig config = {
        6,
        6,
        1,
        app_context ->config.package_path + "/shaders/SetImage_vk_vert.spv",
        "",
        app_context ->config.package_path + "/shaders/SetImage_vk_frag.spv",
        TopologyType::TriangleList,
    };


    Renderable::init(config,app_context);

    width = img_width;
    height = img_height;

    create_texture_image_(width,height);  
    create_texture_image_view_(); 
    create_texture_sampler_(); 

    Renderable::init_render_resources();  

    update_vertex_buffer_();  
    update_index_buffer_();  
}



void SetImage::create_texture_image_(int width, int height) {
        
    VkDeviceSize image_size = (int)(width * height * 4);

    create_image(width,height, VK_FORMAT_R8G8B8A8_UNORM, VK_IMAGE_TILING_OPTIMAL, VK_IMAGE_USAGE_TRANSFER_DST_BIT | VK_IMAGE_USAGE_SAMPLED_BIT, VK_MEMORY_PROPERTY_DEVICE_LOCAL_BIT, texture_image_, texture_image_memory_,app_context_->device,app_context_->physical_device);

    transition_image_layout(texture_image_, VK_FORMAT_R8G8B8A8_UNORM, VK_IMAGE_LAYOUT_UNDEFINED, VK_IMAGE_LAYOUT_TRANSFER_DST_OPTIMAL,app_context_->command_pool,app_context_->device,app_context_->graphics_queue);
    transition_image_layout(texture_image_, VK_FORMAT_R8G8B8A8_UNORM, VK_IMAGE_LAYOUT_TRANSFER_DST_OPTIMAL, VK_IMAGE_LAYOUT_SHADER_READ_ONLY_OPTIMAL,app_context_->command_pool,app_context_->device,app_context_->graphics_queue);

    if(app_context_->config.ti_arch == ARCH_CUDA){
        VkMemoryRequirements mem_requirements;
        vkGetImageMemoryRequirements(app_context_->device, texture_image_, &mem_requirements);

        auto handle = get_device_mem_handle(texture_image_memory_,app_context_->device);
        hipExternalMemory_t external_mem = import_vk_memory_object_from_handle(handle,mem_requirements.size,true);

        texture_surface_ = (uint64_t)get_image_surface_object_of_external_memory(external_mem,width,height);
    }
    create_buffer(image_size, VK_BUFFER_USAGE_TRANSFER_SRC_BIT, VK_MEMORY_PROPERTY_HOST_VISIBLE_BIT | VK_MEMORY_PROPERTY_HOST_COHERENT_BIT, staging_buffer_, staging_buffer_memory_,app_context_->device,app_context_->physical_device);
    
}



void SetImage::create_texture_image_view_() {
    texture_image_view_ = create_image_view(texture_image_, VK_FORMAT_R8G8B8A8_UNORM, VK_IMAGE_ASPECT_COLOR_BIT,app_context_->device);
}

void SetImage::create_texture_sampler_() {
    VkPhysicalDeviceProperties properties{};
    vkGetPhysicalDeviceProperties(app_context_->physical_device, &properties);

    VkSamplerCreateInfo sampler_info{};
    sampler_info.sType = VK_STRUCTURE_TYPE_SAMPLER_CREATE_INFO;
    sampler_info.magFilter = VK_FILTER_LINEAR;
    sampler_info.minFilter = VK_FILTER_LINEAR;
    sampler_info.addressModeU = VK_SAMPLER_ADDRESS_MODE_REPEAT;
    sampler_info.addressModeV = VK_SAMPLER_ADDRESS_MODE_REPEAT;
    sampler_info.addressModeW = VK_SAMPLER_ADDRESS_MODE_REPEAT;
    sampler_info.anisotropyEnable = VK_TRUE;
    sampler_info.maxAnisotropy = properties.limits.maxSamplerAnisotropy;
    sampler_info.borderColor = VK_BORDER_COLOR_INT_OPAQUE_BLACK;
    sampler_info.unnormalizedCoordinates = VK_FALSE;
    sampler_info.compareEnable = VK_FALSE;
    sampler_info.compareOp = VK_COMPARE_OP_ALWAYS;
    sampler_info.mipmapMode = VK_SAMPLER_MIPMAP_MODE_LINEAR;

    if (vkCreateSampler(app_context_->device, &sampler_info, nullptr, &texture_sampler_) != VK_SUCCESS) {
        throw std::runtime_error("failed to create texture sampler!");
    }
}

void SetImage::update_vertex_buffer_(){
    const std::vector<Vertex> vertices = {
        {{-1.f,-1.f,0.f}, {0.f,0.f,1.f},  {0.f,1.f},{1.f,1.f,1.f}},
        {{-1.f,1.f,0.f}, {0.f,0.f,1.f}, {0.f,0.f},{1.f,1.f,1.f}},
        {{1.f,1.f,0.f}, {0.f,0.f,1.f}, {1.f,0.f},{1.f,1.f,1.f}},

        {{ -1.f,-1.f,0.f},{0.f,0.f,1.f}, { 0.f,1.f},{1.f,1.f,1.f}},
        {{ 1.f,1.f,0.f}, {0.f,0.f,1.f}, {1.f,0.f},{1.f,1.f,1.f}},
        {{ 1.f,-1.f,0.f},{0.f,0.f,1.f},  {1.f,1.f},{1.f,1.f,1.f}},
    };

    {
        MappedMemory mapped_vbo(app_context_->device, staging_vertex_buffer_memory_ , config_.vertices_count * sizeof(Vertex));
        memcpy(mapped_vbo.data, vertices.data(), (size_t) config_.vertices_count * sizeof(Vertex));
    }

    copy_buffer(staging_vertex_buffer_, vertex_buffer_, config_.vertices_count * sizeof(Vertex), app_context_ -> command_pool, app_context_ -> device, app_context_ -> graphics_queue) ;

}

void SetImage::update_index_buffer_() {
    const std::vector<uint32_t> indices = {
        0, 1, 2, 3,4,5,
    };
    {
        MappedMemory mapped_ibo(app_context_->device, staging_index_buffer_memory_ , config_.indices_count * sizeof(int));
        memcpy(mapped_ibo.data, indices.data(), (size_t) config_.indices_count * sizeof(int));
    }
    
    copy_buffer(staging_index_buffer_, index_buffer_, config_.indices_count * sizeof(int), app_context_ -> command_pool, app_context_ -> device, app_context_ -> graphics_queue) ;

}


void SetImage::create_descriptor_set_layout()  {
    VkDescriptorSetLayoutBinding ubo_layout_binding{};
    ubo_layout_binding.binding = 0;
    ubo_layout_binding.descriptorCount = 1;
    ubo_layout_binding.descriptorType = VK_DESCRIPTOR_TYPE_UNIFORM_BUFFER;
    ubo_layout_binding.pImmutableSamplers = nullptr;
    ubo_layout_binding.stageFlags = VK_SHADER_STAGE_VERTEX_BIT;

    VkDescriptorSetLayoutBinding sampler_layout_binding{};
    sampler_layout_binding.binding = 1;
    sampler_layout_binding.descriptorCount = 1;
    sampler_layout_binding.descriptorType = VK_DESCRIPTOR_TYPE_COMBINED_IMAGE_SAMPLER;
    sampler_layout_binding.pImmutableSamplers = nullptr;
    sampler_layout_binding.stageFlags = VK_SHADER_STAGE_FRAGMENT_BIT;

    std::array<VkDescriptorSetLayoutBinding, 2> bindings = {ubo_layout_binding, sampler_layout_binding};
    VkDescriptorSetLayoutCreateInfo layout_info{};
    layout_info.sType = VK_STRUCTURE_TYPE_DESCRIPTOR_SET_LAYOUT_CREATE_INFO;
    layout_info.bindingCount = static_cast<uint32_t>(bindings.size());
    layout_info.pBindings = bindings.data();

    if (vkCreateDescriptorSetLayout(app_context_->device, &layout_info, nullptr, &descriptor_set_layout_) != VK_SUCCESS) {
        throw std::runtime_error("failed to create descriptor set layout!");
    }
}

void SetImage::create_descriptor_sets()  {
    std::vector<VkDescriptorSetLayout> layouts(app_context_->get_swap_chain_size(), descriptor_set_layout_);

    VkDescriptorSetAllocateInfo alloc_info{};
    alloc_info.sType = VK_STRUCTURE_TYPE_DESCRIPTOR_SET_ALLOCATE_INFO;
    alloc_info.descriptorPool = descriptor_pool_;
    alloc_info.descriptorSetCount = app_context_->get_swap_chain_size();
    alloc_info.pSetLayouts = layouts.data();

    descriptor_sets_.resize(app_context_->get_swap_chain_size());

    if (vkAllocateDescriptorSets(app_context_->device, &alloc_info, descriptor_sets_.data() ) != VK_SUCCESS) {
        throw std::runtime_error("failed to allocate descriptor sets!");
    }

    for (size_t i = 0; i < app_context_->get_swap_chain_size(); i++) {
        VkDescriptorBufferInfo buffer_info{};
        buffer_info.buffer = uniform_buffers_[i] ;
        buffer_info.offset = 0;
        buffer_info.range = config_.ubo_size;

        VkDescriptorImageInfo image_info{};
        image_info.imageLayout = VK_IMAGE_LAYOUT_SHADER_READ_ONLY_OPTIMAL;
        image_info.imageView = texture_image_view_;
        image_info.sampler = texture_sampler_;

        std::array<VkWriteDescriptorSet, 2> descriptor_writes{};

        descriptor_writes[0].sType = VK_STRUCTURE_TYPE_WRITE_DESCRIPTOR_SET;
        descriptor_writes[0].dstSet = descriptor_sets_[i] ;
        descriptor_writes[0].dstBinding = 0;
        descriptor_writes[0].dstArrayElement = 0;
        descriptor_writes[0].descriptorType = VK_DESCRIPTOR_TYPE_UNIFORM_BUFFER;
        descriptor_writes[0].descriptorCount = 1;
        descriptor_writes[0].pBufferInfo = &buffer_info;

        descriptor_writes[1].sType = VK_STRUCTURE_TYPE_WRITE_DESCRIPTOR_SET;
        descriptor_writes[1].dstSet = descriptor_sets_[i] ;
        descriptor_writes[1].dstBinding = 1;
        descriptor_writes[1].dstArrayElement = 0;
        descriptor_writes[1].descriptorType = VK_DESCRIPTOR_TYPE_COMBINED_IMAGE_SAMPLER;
        descriptor_writes[1].descriptorCount = 1;
        descriptor_writes[1].pImageInfo = &image_info;

        vkUpdateDescriptorSets(app_context_->device, static_cast<uint32_t>(descriptor_writes.size()), descriptor_writes.data(), 0, nullptr);
    }

    
}

void SetImage::cleanup() {
    Renderable::cleanup();

    vkDestroySampler(app_context_->device, texture_sampler_, nullptr);
    vkDestroyImageView(app_context_->device, texture_image_view_, nullptr);

    vkDestroyImage(app_context_->device, texture_image_, nullptr);
    vkFreeMemory(app_context_->device, texture_image_memory_, nullptr);

    vkDestroyBuffer(app_context_->device, staging_buffer_, nullptr);
    vkFreeMemory(app_context_->device, staging_buffer_memory_, nullptr);
}


}//namespace vulkan